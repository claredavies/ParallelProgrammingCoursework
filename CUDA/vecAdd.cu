
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int    NoofReal;
int    NoofRand;

int totaldegrees = 360;
int binsperdegree = 4;
float *h_real_rasc, *h_real_decl;
float *h_rand_rasc, *h_rand_decl;

long int CPUMemory = 0L;

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *real_rasc, float *real_decl, float *rand_rasc, float *rand_decl, int *histogram_DD
    ,int *histogram_DR, int *histogram_RR, int n)
{
    // Get our global thread ID
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    printf("id:   %d    N:  %d \n",idx,n);
    int j;
    if (idx < n) {
         for(j =0; j < n; ++j) {
            atomicAdd(&(histogram_DR[idx]),(int)1);
         }
     }
}
 
int main( int argc, char* argv[] )
{
    int readdata(char *argv1, char *argv2);
    // Size of vectors
    int n = 10;
    //int readdata(char *argv1, char *argv2);

    //Host output vector
    int *h_histogram_DR, *h_histogram_DD, *h_histogram_RR;

    if ( readdata(argv[1], argv[2]) != 0 ) return(-1);

    // Device input vectors
    float *d_real_rasc;
    float *d_real_decl;
    float *d_rand_rasc;
    float *d_rand_decl;

    //Device output vector
    int *d_histogram_DR, *d_histogram_DD, *d_histogram_RR;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
    size_t size_long_int = n*sizeof(long int);
    size_t size_int = n*sizeof(int);
 
    // Allocate memory for each vector on host
    h_real_rasc = (float*)malloc(bytes);
    h_real_decl = (float*)malloc(bytes);
    h_rand_rasc = (float*)malloc(bytes);
    h_rand_decl = (float*)malloc(bytes);

    h_histogram_DD = (int*)malloc(size_int);
    h_histogram_DR = (int*)malloc(size_int);
    h_histogram_RR = (int*)malloc(size_int);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_real_rasc, bytes);
    hipMalloc(&d_real_decl, bytes);
    hipMalloc(&d_rand_rasc, bytes);
    hipMalloc(&d_rand_decl, bytes);
    hipMalloc(&d_histogram_DD, size_int );
    hipMalloc(&d_histogram_DR, size_int );
    hipMalloc(&d_histogram_RR, size_int );
 
    if ( readdata(argv[1], argv[2]) != 0 ) return(-1);

    // Copy host vectors to device
    hipMemcpy(d_real_rasc, h_real_rasc, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_real_decl, h_real_decl, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_rand_rasc, h_rand_rasc, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_rand_decl, h_rand_decl, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_real_rasc, d_real_decl, d_rand_rasc, d_rand_decl, d_histogram_DD, d_histogram_DR
     , d_histogram_RR, n);
 
    // Copy array back to host
    hipMemcpy( h_histogram_DD, d_histogram_DD, size_int, hipMemcpyDeviceToHost );
    hipMemcpy( h_histogram_DR, d_histogram_DR, size_int, hipMemcpyDeviceToHost );
    hipMemcpy( h_histogram_RR, d_histogram_RR, size_int, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum_DD = 0;
    double sum_DR = 0;
    double sum_RR = 0;
    int i;
    for(i=0; i<n; i++) {
         sum_DD += h_histogram_DD[i];
         sum_DR += h_histogram_DR[i];
         sum_RR += h_histogram_RR[i];
    }
    printf("final result DD: %f  final result DR: %f  final result RR: %f\n", sum_DD,sum_DR,sum_RR);
 
    // Release device memory
    hipFree(d_real_rasc);
    hipFree(d_real_decl);
    hipFree(d_rand_rasc);
    hipFree(d_rand_decl);
    hipFree(d_histogram_DD);
    hipFree(d_histogram_DR);
    hipFree(d_histogram_RR);

 
    // Release host memory
    free(h_real_rasc);
    free(h_real_decl);
    free(h_rand_rasc);
    free(h_rand_decl);
    free(h_histogram_DD);
    free(h_histogram_DR);
    free(h_histogram_RR);
 
    return 0;
}

int readdata(char *argv1, char *argv2)
{
  int    i,linecount;
  char   inbuf[80];
  double ra, dec, dpi;
  FILE  *infil;

  printf("   Assuming data is in arc minutes!\n");
                          // phi   = ra/60.0 * dpi/180.0;
                          // theta = (90.0-dec/60.0)*dpi/180.0;
                          // otherwise use
                          // phi   = ra * dpi/180.0;
                          // theta = (90.0-dec)*dpi/180.0;

  dpi = acos(-1.0);
  infil = fopen(argv1,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv1);return(-1);}

  linecount =0;
  while ( fgets(inbuf,80,infil) != NULL ) ++linecount;
  rewind(infil);

  printf("   %s contains %d galaxies\n",argv1, linecount-1);

  NoofReal = linecount-1;

  if ( NoofReal != 100000 ) {printf("Incorrect number of galaxies\n");return(1);}

  h_real_rasc = (float *)calloc(NoofReal,sizeof(float));
  h_real_decl = (float *)calloc(NoofReal,sizeof(float));
  CPUMemory += 2L*NoofReal*sizeof(float);

  fgets(inbuf,80,infil);
  sscanf(inbuf,"%d",&linecount);
  if ( linecount != 100000 ) {printf("Incorrect number of galaxies\n");return(1);}

  i = 0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 )
         {
         printf("   Cannot read line %d in %s\n",i+1,argv1);
         fclose(infil);
         return(-1);
         }
      h_real_rasc[i] = (float)( ra/60.0*dpi/180.0);
      h_real_decl[i] = (float)(dec/60.0*dpi/180.0);
      ++i;
      }

  fclose(infil);

  if ( i != NoofReal )
      {
      printf("   Cannot read %s correctly\n",argv1);
      return(-1);
      }

  infil = fopen(argv2,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv2);return(-1);}

  linecount =0;
  while ( fgets(inbuf,80,infil) != NULL ) ++linecount;
  rewind(infil);

  printf("   %s contains %d galaxies\n",argv2, linecount-1);

  NoofRand = linecount-1;
  if ( NoofRand != 100000 ) {printf("Incorrect number of random galaxies\n");return(1);}

  h_rand_rasc = (float *)calloc(NoofRand,sizeof(float));
  h_rand_decl = (float *)calloc(NoofRand,sizeof(float));
  CPUMemory += 2L*NoofRand*sizeof(float);

  fgets(inbuf,80,infil);
  sscanf(inbuf,"%d",&linecount);
  if ( linecount != 100000 ) {printf("Incorrect number of random galaxies\n");return(1);}

  i =0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 )
         {
         printf("   Cannot read line %d in %s\n",i+1,argv2);
         fclose(infil);
         return(-1);
         }
      h_rand_rasc[i] = (float)( ra/60.0*dpi/180.0);
      h_rand_decl[i] = (float)(dec/60.0*dpi/180.0);
      ++i;
      }

  fclose(infil);

  if ( i != NoofReal )
      {
      printf("   Cannot read %s correctly\n",argv2);
      return(-1);
      }

  return(0);
}